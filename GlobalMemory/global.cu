#include "hip/hip_runtime.h"
	#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>
#include <chrono>

#include "functionImage.h"
#include ""


using namespace std::chrono;
using namespace std;

#define maskCols 5
#define maskRows 5


__global__ void globalMemoryConvolution(float * InputImageData, const float *__restrict__ kernel,
		float* outputImageData, int channels, int width, int height){

	float Pvalue = 0;
	
	int col = threadIdx.x + blockIdx.x * blockDim.x; //number of threads along x axis
	int row = threadIdx.y + blockIdx.y * blockDim.y; //number of threads along y axis
	int maskRowsRadius = maskRows/2;
	int maskColsRadius = maskCols/2;

	for (int k = 0; k < channels; k++){    
		if(row < height && col < width ){
			Pvalue = 0;
			int startRow = row - maskRowsRadius;    
			int startCol = col - maskColsRadius;	
			
			for(int i = 0; i < maskRows; i++){	

				for(int j = 0; j < maskCols; j++){	

					int currentRow = startRow + i;	
					int currentCol = startCol + j;	

					if(currentRow > -1 && currentRow < height && currentCol > -1 && currentCol < width){ // Check the unused threads

							Pvalue += InputImageData[(currentRow * width + currentCol )*channels + k] *
										kernel[i * maskRows + j];
					}
					else Pvalue = 0;
				}

			}
			outputImageData[(row* width + col) * channels + k] = Pvalue;
		}
			
	}
	
}

void imageConvolutionGlobalMemory(const char* inputfilepath, const char* outputfilepath ){

	int imgChannels;
    int imgWidth;
    int imgHeight;
    Image* imgInput;
    Image* imgOutput;
    float* hostInputImageData;
    float* hostOutputImageData;
    float* deviceInputImageData;
    float* deviceOutputImageData;
    float* deviceMaskData;
    float hostMaskData[maskRows * maskCols] = {
			0.06, 0.06, 0.06, 0.06, 0.06,
			0.06, 0.06, 0.06, 0.06, 0.06,
			0.06, 0.06, 0.06, 0.06, 0.06,
			0.06, 0.06, 0.06, 0.06, 0.06,
			0.06, 0.06, 0.06, 0.06, 0.06
    };
    
    imgInput = import_PPM(inputfilepath);
    imgWidth = img_getWidth(imgInput);
    imgHeight = img_getHeight(imgInput);
    imgChannels = img_getChannels(imgInput);

    imgOutput = Image_new(imgWidth, imgHeight, imgChannels);
    
    hostInputImageData = img_getData(imgInput);
    hostOutputImageData = img_getData(imgOutput);

	hipDeviceReset();
	hipMalloc((void **) &deviceInputImageData, imgWidth * imgHeight *
			imgChannels * sizeof(float));
	hipMalloc((void **) &deviceOutputImageData, imgWidth * imgHeight *
			imgChannels * sizeof(float));
	hipMalloc((void **) &deviceMaskData, maskRows * maskCols
			* sizeof(float));
	hipMemcpy(deviceInputImageData, hostInputImageData,
			imgWidth * imgHeight * imgChannels * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(deviceMaskData, hostMaskData,
				maskRows * maskCols * sizeof(float),
				hipMemcpyHostToDevice);

    dim3 dimGrid(ceil((float)imgWidth / 16),
        ceil((float)imgHeight / 16));  			// N	 thread blocks
        
	dim3 dimBlock(16, 16, 1); 					//16x16 thread per block
    
    cout << "CONVOLUTION GLOBAL MEMORY" << endl;
    cout << "Image dimensions : " << imgWidth << "x" << imgHeight << " , Channels : " << imgChannels << endl;
    high_resolution_clock::time_point start = high_resolution_clock::now();

								// N x 256 threads
    globalMemoryConvolution <<<dimGrid, dimBlock >>>(deviceInputImageData, deviceMaskData, deviceOutputImageData,
        imgChannels, imgWidth, imgHeight);

    high_resolution_clock::time_point end = high_resolution_clock::now();
    chrono::duration<double> duration = end - start;
    cout << "Time: " << duration.count() * 1000 << endl;
    cout << "----------------------------------" << endl;

    // copies "count" bytes from the memory area pointed to by deviceOutputImageData to the memory area pointer to by hostOutputImageData
    hipMemcpy(hostOutputImageData, deviceOutputImageData,static_cast<unsigned long long>(imgWidth) * imgHeight *
        imgChannels * sizeof(float), hipMemcpyDeviceToHost);

    write_image(outputfilepath, imgOutput);

	//Fills the first "count" bytes of the memory area pointed to by "deviceInputImageData" with the constant byte value "0"
	size_t count = static_cast<unsigned long long>(imgWidth) * imgHeight * imgChannels * sizeof(float);
	
	hipMemset(deviceInputImageData,0,count);
	hipMemset(deviceOutputImageData,0,count);
	hipMemset(deviceMaskData,0,maskRows * maskCols
				* sizeof(float));
    
    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceMaskData);
    
    Image_delete(imgInput);
    Image_delete(imgOutput);
    
}

int main() {

	imageConvolutionGlobalMemory("/home/aventuri/progetto/globalmemory/photoSD.ppm","/home/aventuri/progetto/globalmemory/resultSDGM.ppm");
	imageConvolutionGlobalMemory("/home/aventuri/progetto/globalmemory/photoHD1.ppm","/home/aventuri/progetto/globalmemory/resultHD1GM.ppm");
	imageConvolutionGlobalMemory("/home/aventuri/progetto/globalmemory/photoHD2.ppm","/home/aventuri/progetto/globalmemory/resultHD2GM.ppm");
	imageConvolutionGlobalMemory("/home/aventuri/progetto/globalmemory/photo4K.ppm","/home/aventuri/progetto/globalmemory/result4KGM.ppm");

}

